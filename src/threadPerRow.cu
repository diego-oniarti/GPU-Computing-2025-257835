#include "hip/hip_runtime.h"
#include "common.h"
#include "timing.h"
#include "matrix.h"
#include <stdio.h>

__global__
void mult_per_row_kelner(data_t *vals, int *xs, int *ys, 
        data_t *vec, data_t *ret,
        int cols, int rows) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < rows) {
        data_t acc = 0;
        for (int i=ys[tid]; i<ys[tid+1]; i++) {
            acc += vals[i] * vec[xs[i]];
        }
        ret[tid] = acc;
    }
}

data_t* mult_per_row(MAT_CSR *csr, data_t *ones, int maxThreads) {
    // Create as many threads as matrix rows
    int n_threads = csr->nrows;
    // Enough blocks to accomodate the threads
    int n_blocks = ceil((float)n_threads / maxThreads);

    // Put the data into managed memory to make it accessible by the GPU
    data_t *vals, *vec, *ret;
    int *xs, *ys;
    hipMallocManaged(&vals, sizeof(data_t)*csr->nvals);
    hipMemcpy(vals, csr->vals, sizeof(data_t)*csr->nvals, hipMemcpyHostToDevice);
    hipMallocManaged(&vec, sizeof(data_t)*COLS);
    hipMemcpy(vec, ones, sizeof(data_t)*COLS, hipMemcpyHostToDevice);
    hipMallocManaged(&ret, sizeof(data_t)*ROWS);
    hipMallocManaged(&xs, sizeof(int)*csr->nvals);
    hipMemcpy(xs, csr->xs, sizeof(int)*csr->nvals, hipMemcpyHostToDevice);
    hipMallocManaged(&ys, sizeof(data_t)*(ROWS+1));
    hipMemcpy(ys, csr->ys, sizeof(data_t)*(ROWS+1), hipMemcpyHostToDevice);

    // Events and array for timing
    double times[RUNS];
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int r=-PRERUNS; r<RUNS; r++) {
        hipEventRecord(start);
        mult_per_row_kelner<<<n_blocks, maxThreads>>>(vals, xs, ys, vec, ret, COLS, ROWS);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        hipDeviceSynchronize();

        if (DOPRINTSINGLE) printf("--- Elapsed time: %lf\n", milliseconds);
        if (r>=0) { // Preruns
            times[r] = milliseconds;
        }
    }
    print_timing(times, RUNS, csr->nvals*2);

    hipFree(vals);
    hipFree(vec);
    hipFree(xs);
    hipFree(ys);

    return ret;
}
