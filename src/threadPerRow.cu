#include "hip/hip_runtime.h"
#include "common.h"
#include "timing.h"
#include "matrix.h"
#include <stdio.h>

__global__
void mult_per_row_kelner(data_t *vals, int *xs, int *ys, 
        data_t *vec, data_t *ret,
        int cols, int rows) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < rows) {
        data_t acc = 0;
        for (int i=ys[tid]; i<ys[tid+1]; i++) {
            acc += vals[i] * vec[xs[i]];
        }
        ret[tid] = acc;
    }
}

data_t* mult_per_row(MAT_CSR *csr, data_t *ones, int maxThreads) {
    int ROWS = csr->nrows;
    int COLS = csr->ncols;

    // Create as many threads as matrix rows
    int n_threads = csr->nrows;
    // Enough blocks to accomodate the threads
    int n_blocks = ceil((float)n_threads / maxThreads);

    // Put the data into managed memory to make it accessible by the GPU
    data_t *vals, *vec, *ret;
    int *xs, *ys;
    hipMalloc(&vals, sizeof(data_t)*csr->nvals);
    hipMemcpy(vals, csr->vals, sizeof(data_t)*csr->nvals, hipMemcpyHostToDevice);
    hipMalloc(&vec, sizeof(data_t)*COLS);
    hipMemcpy(vec, ones, sizeof(data_t)*COLS, hipMemcpyHostToDevice);
    hipMallocManaged(&ret, sizeof(data_t)*ROWS);
    hipMalloc(&xs, sizeof(int)*csr->nvals);
    hipMemcpy(xs, csr->xs, sizeof(int)*csr->nvals, hipMemcpyHostToDevice);
    hipMalloc(&ys, sizeof(int)*(ROWS+1));
    hipMemcpy(ys, csr->ys, sizeof(int)*(ROWS+1), hipMemcpyHostToDevice);

    // Events and array for timing
    double times[RUNS];
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int r=-PRERUNS; r<RUNS; r++) {
        hipEventRecord(start);
        mult_per_row_kelner<<<n_blocks, maxThreads>>>(vals, xs, ys, vec, ret, COLS, ROWS);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        hipDeviceSynchronize();

        if (DOPRINTSINGLE) printf("--- Elapsed time: %lf\n", milliseconds);
        if (r>=0) { // Preruns
            times[r] = milliseconds;
        }
    }
    print_timing(times, RUNS, csr->nvals*2);

    hipFree(vals);
    hipFree(vec);
    hipFree(xs);
    hipFree(ys);

    return ret;
}
