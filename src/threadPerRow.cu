#include "hip/hip_runtime.h"
#include "common.h"
#include "timing.h"
#include "matrix.h"
#include <stdio.h>

__global__
void mult_per_row_kelner(data_t *vals, int *xs, int *ys, 
        data_t *vec, data_t *ret,
        int cols, int rows) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < rows) {
        data_t acc = 0;
        for (int i=ys[tid]; i<ys[tid+1]; i++) {
            acc += vals[i] * vec[xs[i]];
        }
        ret[tid] = acc;
    }
}

data_t* mult_per_row(MAT_CSR *csr, data_t *ones) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int maxThreads = prop.maxThreadsPerBlock;

    int n_threads = csr->nrows;
    int n_blocks = ceil((float)n_threads / maxThreads);

    data_t *vals, *vec, *ret;
    int *xs, *ys;
    hipMallocManaged(&vals, sizeof(data_t)*csr->nvals);
    hipMemcpy(vals, csr->vals, sizeof(data_t)*csr->nvals, hipMemcpyHostToDevice);
    hipMallocManaged(&vec, sizeof(data_t)*COLS);
    hipMemcpy(vec, ones, sizeof(data_t)*COLS, hipMemcpyHostToDevice);
    hipMallocManaged(&ret, sizeof(data_t)*ROWS);
    hipMallocManaged(&xs, sizeof(int)*csr->nvals);
    hipMemcpy(xs, csr->xs, sizeof(int)*csr->nvals, hipMemcpyHostToDevice);
    hipMallocManaged(&ys, sizeof(data_t)*(ROWS+1));
    hipMemcpy(ys, csr->ys, sizeof(data_t)*(ROWS+1), hipMemcpyHostToDevice);

    double times[RUNS];
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int r=-PRERUNS; r<RUNS; r++) {
        hipEventRecord(start);
        mult_per_row_kelner<<<n_blocks, maxThreads>>>(vals, xs, ys, vec, ret, COLS, ROWS);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        hipDeviceSynchronize();

        printf("--- Elapsed time: %lf\n", milliseconds);
        if (r>=0) {
            times[r] = milliseconds;
        }
    }
    print_timing(times, RUNS);

    hipFree(vals);
    hipFree(vec);
    hipFree(xs);
    hipFree(ys);

    return ret;
}
