#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "matrix.h"

/*
 * Generates a sparse matrix where each element has a
 * uniform probability `p` of being nonzero.
 * The values are in the range [0-10] and have 2 decimal points
 */
data_t* get_sparse_matrix(int rows, int cols, float p) {
    data_t *ret = (data_t*)malloc(sizeof(data_t) * cols * rows);
    for (int y=0; y<rows; y++) {
        for (int x=0; x<cols; x++) {
            ret[y*cols + x] = (rand()%100 < p*100) ? (data_t)(rand()%1000)/100. : 0;
            //printf("%f ", ret[y*rows + x]);
        }
    }

    return ret;
}

/*
 * Generates a vector of the given size filled with ones
 */
data_t* get_ones(int n) {
    data_t *ret = (data_t*)malloc(sizeof(data_t) * n);
    for (int i=0; i<n; i++) {
        ret[i] = (data_t)(rand()%1000)/100.;
    }

    return ret;
}

data_t* get_random_vec(int n) {
    data_t *ret = (data_t*)malloc(sizeof(data_t) * n);
    
    for (int i=0; i<n; i++) {
        ret[i] = 1;
    }

    return ret;
}

int count_non_zeros(data_t *mat, int rows, int cols) {
    int acc = 0;
    for (int y=0; y<rows; y++) {
        for (int x=0; x<cols; x++) {
            if (mat[y*cols+x]!=0) {
                acc++;
            }
        }
    }
    return acc;
}

/**
 * Populates a CSR matrix with the values from a matrix
 */
void mat_to_CSR(MAT_CSR *csr, data_t *mat, int cols, int rows) {
    int nvals  = count_non_zeros(mat, rows, cols);
    csr->nvals = nvals;
    csr->nrows = rows;
    csr->ncols = cols;
    csr->vals  = (data_t*)malloc(sizeof(data_t) * nvals);
    csr->xs    = (int*)malloc(sizeof(int) * nvals);
    csr->ys    = (int*)malloc(sizeof(int) * (rows+1));
    csr->ys[0]=0;
    int n=0;
    for (int y=0; y<rows; y++) {
        for (int x=0; x<cols && n<nvals; x++) {
            if (mat[y*cols+x]!=0) {
                csr->vals[n] = mat[y*cols+x];
                csr->xs[n] = x;
                n++;
            }
        }
        csr->ys[y+1] = n;
    }
}

void destroy_CSR(MAT_CSR *csr) {
    free(csr->vals);
    free(csr->xs);
    free(csr->ys);
}

void print_array(data_t *arr, int n) {
    for (int i=0; i<n; i++) {
        printf("%.2f ", arr[i]);
    }
    printf("\n");
}

void print_array_i(int *arr, int n) {
    for (int i=0; i<n; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}

void print_CSR(MAT_CSR *csr) {
    printf("Vals:  |");
    print_array(csr->vals, csr->nvals);
    printf("Xs:    |");
    print_array_i(csr->xs, csr->nvals);
    printf("Ys:    |");
    print_array_i(csr->ys, csr->nrows+1);
}

bool check_equal(data_t *m1, data_t *m2, int n) {
    for (int i=0; i<n; i++) {
        if (m1[i] != m2[i]) return false;
    }
    return true;
}

void assert_correct(data_t *m1, data_t *m2, int n) {
    data_t maxErr = 0;
    int n_errors = 0;
    for (int i=0; i<n; i++) {
        if (m1[i] != m2[i]) {
            data_t error = abs(m2[i]-m1[i]);
            if (error>maxErr) maxErr=error;
            n_errors++;
        }
    }
    if (maxErr != 0) {
        fprintf(stderr, "!!! - Num errors: %d | Max error: %.17g\n", n_errors, maxErr);
        // print error as binary
        long long *errInt = (long long*)&maxErr;
        for (int i=63; i>=0; i--) {
            long long b = (*errInt) & ((long long)1<<i);
            if (b==0) {
                fprintf(stderr, "0");
            }else{
                fprintf(stderr, "1");
            }
        }
        fprintf(stderr, "\n");
        
        // exit(1);
    }
}

void read_mtx(MAT_CSR *mat, const char *path) {
    FILE *file = fopen(path, "r");
    if (!file) return;

    char *line = (char*)malloc(sizeof(char)*255);
    size_t len = 255;

    int rows, cols, nonzeros;
    while (getline(&line, &len, file) != -1) {
        if (len==0 || line[0]=='%') continue;
        sscanf(line, "%d %d %d", &rows, &cols, &nonzeros);
        break;
    }
    printf("rows %d\ncols %d\nnonzeros %d\n", rows, cols, nonzeros);

    fpos_t pos;  // Declare a position holder
    fgetpos(file, &pos);  // Save current position

    mat->nvals = nonzeros;
    mat->ncols = cols;
    mat->nrows = rows;
    mat->vals  = (data_t*)malloc(sizeof(data_t) * nonzeros);
    mat->xs    = (int*)malloc(sizeof(int) * nonzeros);
    mat->ys    = (int*)calloc(rows+1, sizeof(int));
    int *row_counter = (int*)calloc(rows, sizeof(int));

    // First cycle to count the number of elements on each row
    for (int i=0; i<nonzeros; i++) {
        int y=0, x=0;
        data_t val=0;
        getline(&line, &len, file);
        sscanf(line, "%d %d %lf", &y, &x, &val);
        y--;
        x--;
        mat->ys[y+1]++;
    }
    // Incremental sum of the row pointer
    for (int i=1; i<=rows; i++) {
        mat->ys[i] += mat->ys[i-1];
    }

    // Reset the position in the file to read the lines again
    fsetpos(file, &pos);
    for (int i=0; i<nonzeros; i++) {
        int y=0, x=0;
        data_t val=0;
        getline(&line, &len, file);
        sscanf(line, "%d %d %lf", &y, &x, &val);
        y--;
        x--;

        //int p = mat->ys[y]+(row_counter[y]++);
        int base = mat->ys[y];
        int holding_x = x;
        data_t holding_val = val;
        for (int i=0; i<row_counter[y]; i++) {
            if ( holding_x < mat->xs[base+i] ) {
                int tmp_x = mat->xs[base+i];
                mat->xs[base+i] = holding_x;
                holding_x = tmp_x;

                data_t tmp_val = mat->vals[base+i];
                mat->vals[base+i] = holding_val;
                holding_val = tmp_val;
            }
        }
        mat->xs[base+row_counter[y]] = holding_x;
        mat->vals[base+row_counter[y]] = holding_val;
        row_counter[y] += 1;
    }

    fclose(file);
    free(line);
    free(row_counter);
}
