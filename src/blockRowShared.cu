#include "hip/hip_runtime.h"
#include "common.h"
#include "timing.h"
#include "matrix.h"
#include "warpRowShared.h"
#include <stdio.h>

__global__
void kernel_block_row(data_t *vals, int *xs, int *ys, 
        data_t *vec, data_t *ret,
        int cols, int rows) {
    // Thread id
    int tid = threadIdx.x;
    // Block id
    int bid = blockIdx.x;
    // Row: blockid * rows_per_block + warpid
    int row = bid;

    extern __shared__ data_t smem[]; //One shared memory for both buffer and vals
    data_t *buffer = smem;
    //int offset = blockDim.x; // Size of the buffer
    // data_t *shared_vec = smem+offset;

    buffer[tid] = 0;
    if (row < rows) {
        int start = ys[row];
        int end = ys[row+1];

        // The sum of the elements taken by this thread. In case there are more values in
        // a row than threads in a warp
        data_t sum = 0; 
        for (int i=start+tid; i<end; i+=blockDim.x) {
            // The access to the vector is not coalesced
            sum += vals[i] * vec[xs[i]];
        }

        buffer[tid] = sum;
    }

    for (int s=1; s<blockDim.x; s<<=1) {
        __syncthreads();
        if ((tid & ((s<<1)-1)) == 0) {
            buffer[tid] += buffer[tid+s];
        }
    }

    if (tid==0 && row<rows) {
        ret[row] = buffer[tid];
    }
}

data_t* mult_block_row_shared(MAT_CSR *csr, data_t *ones, int threads_per_block) {
    int ROWS = csr->nrows;
    int COLS = csr->ncols;

    int n_blocks = ROWS;

    // Put the data into managed memory to make it accessible by the GPU
    data_t *vals, *vec, *ret;
    int *xs, *ys;
    hipMalloc(&vals, sizeof(data_t)*csr->nvals);
    hipMemcpy(vals, csr->vals, sizeof(data_t)*csr->nvals, hipMemcpyHostToDevice);
    hipMalloc(&vec, sizeof(data_t)*COLS);
    hipMemcpy(vec, ones, sizeof(data_t)*COLS, hipMemcpyHostToDevice);
    hipMallocManaged(&ret, sizeof(data_t)*ROWS);
    hipMalloc(&xs, sizeof(int)*csr->nvals);
    hipMemcpy(xs, csr->xs, sizeof(int)*csr->nvals, hipMemcpyHostToDevice);
    hipMalloc(&ys, sizeof(int)*(ROWS+1));
    hipMemcpy(ys, csr->ys, sizeof(int)*(ROWS+1), hipMemcpyHostToDevice);

    // Events and array for timing
    double times[RUNS];
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int r=-PRERUNS; r<RUNS; r++) {
        hipEventRecord(start);
        size_t buffer_size = sizeof(data_t)*threads_per_block;
        // size_t vec_size = sizeof(data_t)*COLS;
        kernel_block_row<<<n_blocks, threads_per_block, buffer_size>>>(
                vals, xs, ys, vec, ret, COLS, ROWS
                );
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("kernel launch failed: %s\n", hipGetErrorString(err));
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        hipDeviceSynchronize();

        if (DOPRINTSINGLE) printf("--- Elapsed time: %lf\n", milliseconds);
        if (r>=0) { // Preruns
            times[r] = milliseconds;
        }
    }
    print_timing(times, RUNS, csr->nvals*2, csr);

    hipFree(vals);
    hipFree(vec);
    hipFree(xs);
    hipFree(ys);

    return ret;
}
